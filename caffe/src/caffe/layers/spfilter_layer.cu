/* UNC Software �Learned Contextual Feature Reweighting for Image Geo-Localization�
Copyright (C) 2017 The University of North Carolina at Chapel Hill
All rights reserved.
Written by Hyo Jin Kim (hyojin@cs.unc.edu)
*/
#include <vector>

#include "caffe/layers/spfilter_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void SpfilterLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
	//Forward_cpu(bottom, top);
	
	const int selector_index = bottom.size() - 1;
	const Dtype* select_data = bottom[selector_index]->gpu_data();
  // forward all filtered items for all bottoms but the Selector (bottom[last])
  for (int t = 0; t < top.size(); ++t) {
    const Dtype* bottom_data = bottom[t]->gpu_data();
    Dtype* top_data = top[t]->mutable_gpu_data();

	const int num = bottom[t]->shape(0);
	const int channels = bottom[t]->shape(1);
	const int dim = bottom[t]->count() / num;
	const int area = (bottom[t]->shape(2))*(bottom[t]->shape(3));

    for (int n = 0; n < num; ++n) {
		const Dtype* curr_select_data = select_data + n*area;
		const Dtype* curr_bottom_data = bottom_data + n*dim;
		Dtype* curr_top_data = top_data + n*dim;

		for (int c = 0; c < channels; ++c) {
			caffe_gpu_mul(area, curr_select_data, curr_bottom_data + c*area, curr_top_data + c*area);
		}
    }
  }
  
}

template <typename Dtype>
void SpfilterLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	//Backward_cpu(top, propagate_down, bottom);
	
	const int selector_index = bottom.size() - 1;
	const Dtype* select_data = bottom[selector_index]->gpu_data();
	Dtype* select_diff = bottom[selector_index]->mutable_gpu_diff();

	Dtype* temp_data = temp_.mutable_gpu_data();

	caffe_gpu_set(bottom[selector_index]->count(), Dtype(0.0), select_diff);

  for (int t = 0; t < top.size(); t++) {
	  const Dtype* top_data = top[t]->gpu_data();
	  const Dtype* bottom_data = bottom[t]->gpu_data();
	  Dtype* top_diff = top[t]->mutable_gpu_diff();
	  Dtype* bottom_diff = bottom[t]->mutable_gpu_diff();

	  const int count = bottom[t]->count();
	  const int num = bottom[t]->shape(0);
	  const int channels = bottom[t]->shape(1);
	  const int dim = count / num;
	  const int area = (bottom[t]->shape(2))*(bottom[t]->shape(3));

	  // if (propagate_down[t])
	  caffe_gpu_mul(count, bottom_data, top_diff, temp_data);

	  for (int n = 0; n < num; n++) {
		  const Dtype* curr_select_data = select_data + n*area;
		  Dtype* curr_select_diff = select_diff + n*area;

		  Dtype* curr_bottom_diff = bottom_diff + n*dim;

		  for (int c = 0; c < channels; ++c) {
			  // for non-selector
			  caffe_copy(area, curr_select_data, curr_bottom_diff + c*area);
			  // for selector
			  caffe_gpu_axpy(area, Dtype(1), temp_data + n*dim + c*area, curr_select_diff);
		  }
	  }

	  // for non-selector
	  caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);

    }
	
  }


INSTANTIATE_LAYER_GPU_FUNCS(SpfilterLayer);

}  // namespace caffe
